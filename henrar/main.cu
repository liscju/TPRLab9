
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

void fun()
{
	const int vector_size = 200;
	double *CPU_A, *CPU_B, *CPU_C;
	double *GPU_A, *GPU_B, *GPU_C;

	CPU_A = new double[vector_size];
	CPU_B = new double[vector_size];
	CPU_C = new double[vector_size];

	for(int i = 0; i < vector_size; i++)
	{
		CPU_A[i] = static_cast<double>(i);
		CPU_B[i] = static_cast<double>(i);
	}
	
	cublasAlloc(vector_size, sizeof(double), (void **) &GPU_A);
	cublasAlloc(vector_size, sizeof(double), (void **) &GPU_B);
	cublasAlloc(vector_size, sizeof(double), (void **) &GPU_C);
	
	hipblasSetVector(vector_size, sizeof(double), CPU_A, 1, GPU_A, 1);
	hipblasSetVector(vector_size, sizeof(double), CPU_B, 1, GPU_B, 1);

	cublasDaxpy(vector_size, 1.0, GPU_A, 1, GPU_B, 1);

	hipblasGetVector(vector_size, sizeof(double), GPU_B, 1, CPU_C, 1);

	for(int i = 0; i < vector_size; i++)
	{
		std::cout << CPU_C[i] << std::endl;
	}

	delete[] CPU_A;
	delete[] CPU_B;
	delete[] CPU_C;
	
	cublasFree(GPU_A);
	cublasFree(GPU_B);
	cublasFree(GPU_C);
}

int main()
{
	cublasInit();
	int version;
	cublasGetVersion(&version);
	std::cout << "Cublas version:" << version << std::endl;
	fun();
	cublasShutdown();
	return 0;
}

