
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

int main()
{
	cublasInit();
	int version;
	cublasGetVersion(&version);
	std::cout << "Cublas version:" << version << std::endl;
	cublasShutdown();
	return 0;
}

